#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <cmath>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <stdio.h>



float recursiveReduce(float *data, int const size)
{
    // terminate check
    if (size == 1) return data[0];

    // renew the stride
    int const stride = size / 2;

    // in-place reduction
    for (int i = 0; i < stride; i++)
    {
        data[i] += data[i + stride];
    }

    // call recursively
    return recursiveReduce(data, stride);
}

__global__ void reduceForLoop (float *g_idata, float *g_odata, unsigned int n)
{

    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    float *idata = g_idata + blockIdx.x * blockDim.x * 8;

    float *ptr = g_idata + idx;
    float tmp = 0;

    // Increment tmp 8 times with values strided by blockDim.x
    for (int i = 0; i < 8; i++) {
        tmp += *ptr;
        ptr += blockDim.x;
    }
    g_idata[idx] = tmp;

    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];

}

int main(int argc, char **argv) {

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    bool bResult = false;

    // initialization
    int size = 1 << 24; // 24 total number of elements to reduce

    printf("    with array size %d  ", size);

    // execution configuration
    int blocksize = 512;   // initial block size

    if(argc > 1)
    {
        blocksize = atoi(argv[1]);   // block size from command line argument
    }

    dim3 block (blocksize, 1);
    dim3 grid  ((size + block.x - 1) / block.x, 1);
    printf("grid %d block %d\n", grid.x, block.x);

    // allocate host memory
    size_t bytes = size * sizeof(float);
    float *h_idata = (float*) malloc(bytes);
    float *h_odata = (float*) malloc(grid.x * sizeof(float));
    float *tmp     = (float*) malloc(bytes);

    // initialize the array
    for (int i = 0; i < size; i++)
    {
        // mask off high 2 bytes to force max number to 255
        h_idata[i] = (float)( rand() & 0xFF );
    }

    memcpy (tmp, h_idata, bytes);

    double iStart, iElaps;
    float gpu_sum = 0.0;

    // allocate device memory
    float *d_idata = NULL;
    float *d_odata = NULL;
    CHECK(hipMalloc((void **) &d_idata, bytes));
    CHECK(hipMalloc((void **) &d_odata, grid.x * sizeof(float)));

    // cpu reduction
    iStart = seconds();
    float cpu_sum = recursiveReduce (tmp, size);
    iElaps = seconds() - iStart;
    printf("cpu reduce      elapsed %f sec cpu_sum: %f\n", iElaps, cpu_sum);

    // exercise 3-5: Implement sum reduction of floats in C.
    // size of 2**24 shows numeric error, 2**22 works.
    // this should be further investigated.

    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = seconds();
    reduceForLoop<<<grid.x/8, block>>>(d_idata, d_odata, size);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 8 * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0.0;

    for (int i = 0; i < grid.x / 8; i++){
        //printf("%d-th gpu_sum BEFORE: %f, h_odata[i]: %f\n", i, gpu_sum, h_odata[i]);
        gpu_sum += h_odata[i];
        //printf("%d-th gpu_sum AFTER:  %f\n", i, gpu_sum);
    }

    printf("gpu Unrolling8  elapsed %f sec gpu_sum: %f <<<grid %d block "
           "%d>>>\n", iElaps, gpu_sum, grid.x / 8, block.x);


    // free host memory
    free(h_idata);
    free(h_odata);

    // free device memory
    CHECK(hipFree(d_idata));
    CHECK(hipFree(d_odata));

    // reset device
    CHECK(hipDeviceReset());

    // check the results
    float diff = (gpu_sum - cpu_sum)*(gpu_sum - cpu_sum);
    bResult = diff < 0.001; // smaller than epsilon

    if(!bResult) printf("Test failed!\n");



    return EXIT_SUCCESS;
}

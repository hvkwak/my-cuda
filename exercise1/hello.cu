
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloWorld(void){
    printf("Hello World frorm GPU!\n");
}

void helloFromGPU(dim3 grid, dim3 block){
    helloWorld<<<grid, block>>>();
    hipDeviceSynchronize();
}

void helloFromCPU(void){
    printf("Hello World from CPU!\n");
}

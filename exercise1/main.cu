#include "../common/common.h"
#include <stdio.h>

void helloFromGPU(dim3 grid, dim3 block);
void helloFromCPU();

int main(void) {

    // hello from CPU
    helloFromCPU();

    // hello from GPU
    dim3 grid(1);
    dim3 block(10);
    helloFromGPU(grid, block);

    // exercise 1-2: see if hipDeviceReset() makes difference
    // resources e.g. memory won't be released if hipDeviceReset() were not called here:
    // CHECK(hipDeviceReset());

    // exercise 1-3: replace hipDeviceReset() with hipDeviceSynchronize();
    // without hipDeviceSynchronize() terminates the host too early, before device prints out the messages.
    CHECK(hipDeviceSynchronize());

    return 0;
}
